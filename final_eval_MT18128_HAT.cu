#include "hip/hip_runtime.h"
//Krishna Bagaria MT18128
//HAT --- Hashed Array Tree

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define size 10000 //initial size of HAT

int count_ints (const char* file_name)
{
	FILE* file = fopen (file_name, "r");
	int i = 0;
	int inputsize = 0;
	fscanf (file, "%d", &i);    
	while (!feof (file))
	{  
		//printf ("%d ", i);
		fscanf (file, "%d", &i);
		inputsize++;
	}
	//printf("\n%d",inputsize);
	fclose (file);
	return inputsize;
}

void read_ints (const char* file_name, int * input)
{
	FILE* file = fopen (file_name, "r");
	int i = 0;
	int inputsize = 0;
	fscanf (file, "%d", &i);    
	while (!feof (file))
	{  
		input[inputsize] = i;
		//printf ("%d ", i);
		inputsize++;
		fscanf (file, "%d", &i);
	}
	//printf("%d",inputsize);
	fclose (file);
}

// method to print the HAT
void print_tree(int ** HAT, int n){
	for (int i = 0; i < n; i++){
		if (HAT[i] != NULL){
			printf("\nBucket %d : ",i);
			for (int j =0; j<n ; j++){
				printf("%d ",HAT[i][j]);
			}
		}
		else {
			printf("\nBucket %d is empty.",i);
		}
	}
}

//CPU method to insert elements in HAT from 'input' array passed as argument
void insert_tree (int ** HAT, int n, int * input, int inputsize) {
	for (int i = 0; i < inputsize; i++){
		int j = i/n;
		if (HAT[j]== NULL){
			HAT[j] = (int *)malloc(sizeof(int) * n);
		}
		int k = i % n;
		HAT[j][k] = input[i];
	}
}

void delete_tree (int ** HAT, int n, int inputsize, int * keys, int keysize) {
	
	for (int l =0; l<keysize; l++){
		for (int i = 0; i < inputsize; i++){
			int j = i/n;
			int k = i % n;
			if (HAT[j][k] == keys[l]) {
				HAT[j][k] = -999999;
			};
		}
	}
	
}

__global__ void delete_gpu (int ** HAT_d, int n, int inputsize, int key) {
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	if (i< inputsize) {
		int j = i/n;
		int k = i % n;
		if (HAT_d[j][k] == key) {
			HAT_d[j][k] = -999999;	
		};
	}
}

void search_tree (int ** HAT, int n, int inputsize, int key) {
	for (int i = 0; i < inputsize; i++){
		int j = i/n;
		int k = i % n;
		if (HAT[j][k] == key) {
			printf ("\n%d found at index %d (Bucket %d, Position %d)", key,i,j,k);	
		};
	}
}

__global__ void search_gpu (int ** HAT_d, int n, int inputsize, int key) {
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	if (i < inputsize){
		int j = i/n;
		int k = i % n;
		if (HAT_d[j][k] == key) {
			printf ("\n%d found at index %d (Bucket %d, Position %d)", key,i,j,k);	
		};
	}
}

//GPU method for inserting elements in HAT
__global__ void insert_gpu (int ** HAT_d, int *input, int inputsize, int n) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	//printf("\n%d",i);
	if(i < inputsize){
	
		int j = i/n;
		int k = i % n;
		printf("\n%d",HAT_d[j][k]);
		HAT_d[j][k] = input[i];
		
	}
}


int main (int argc, const char **argv) {
	int ** HAT;
	int n = sqrt(size);   //calculate size of main array or each leaf
	HAT = (int **)malloc(sizeof(int *) * n);
	
	int inputsize = count_ints ("data/input10000.txt");
	int * input = (int *)malloc(sizeof(int) * inputsize);
	read_ints("data/input10000.txt",input);
	
	/*
	for (int i =0; i<inputsize; i++) {
		printf("%d ", input[i]);
	}
	*/
	
	/*
	printf("Enter no. of elements to be inserted: ");
	scanf("%d",&inputsize);
	
	int * input = (int *)malloc(sizeof(int) * inputsize);
	printf("Enter the elements (integers): ");
	
	for (int i =0; i<inputsize; i++) {
		scanf("%d", &input[i]);
	}
	*/
	int keysize= 100;
	int *keys = (int *)malloc(sizeof(int)*keysize);
	for (int i=0; i<keysize; i++){
		keys[i] = i;
	}
	
	//search_tree(HAT,n,inputsize,2);
	
	const clock_t begin_time = clock();  // measure CPU time for insertion
	insert_tree(HAT, n , input, inputsize);
	printf("\nOutput Tree by inserting from CPU:\n");
	print_tree(HAT,n);
	
	delete_tree(HAT, n, inputsize, keys, keysize);
	float runTime_cpu = (float)( clock() - begin_time ) / CLOCKS_PER_SEC;
	
	
	
	printf("\nOutput Tree by deleting from CPU:\n");
	print_tree(HAT,n);
	
	
	//GPU code starts here
	int * input_d;
	int ** HAT_d;
	
	hipMalloc ((void **)&input_d , sizeof(int) * inputsize);
	hipMalloc ((void ***)&HAT_d , sizeof(int *) * n);
	
	int **support;
	support = (int**) malloc(sizeof(int*)*n);
	hipMemcpy(support, HAT_d, n*sizeof(int*), hipMemcpyDeviceToHost);
	
	for (int i =0; i<n; i++){
		hipMalloc((void**)&support[i],sizeof(int) * n);
	}
	
	hipMemcpy (input_d, input, sizeof(int) * inputsize , hipMemcpyHostToDevice);
	
	int grid_size = (inputsize % 1024) ? ((inputsize/1024) + 1) : (inputsize/1024);
	int block_size = 1024;
	
	const clock_t begin_time1 = clock(); 
	insert_gpu<<<grid_size,block_size>>>(HAT_d,input_d, inputsize, n);
	hipDeviceSynchronize();
	//search_gpu<<<grid_size,block_size>>>(HAT_d,n,inputsize,2);
	
	hipStream_t stream[keysize];
	for (int i =0;i<keysize; i++){
		hipStreamCreate(&stream[i]);
		delete_gpu<<< grid_size,block_size,0,stream[i]>>>(HAT, n, inputsize, keys[i]);
	}
	hipDeviceSynchronize();
	float runTime_gpu1 = (float)( clock() - begin_time1 ) /  CLOCKS_PER_SEC;
	
	hipMemcpy (HAT, HAT_d, sizeof(int*) * n , hipMemcpyDeviceToHost);
	float runTime_gpu2 = (float)( clock() - begin_time1 ) /  CLOCKS_PER_SEC;
	
	printf("\nOutput Tree by deleting from GPU:\n");
	print_tree(HAT,n);
	
	printf("\n\nGPU Kernel Time: %fs\n\n", runTime_gpu1);
	printf("\nGPU Kernel + Memory Transfer Time: %fs\n\n", runTime_gpu2);
	printf("\nTime for inserting(CPU): %fs\n\n", runTime_cpu);
	
	hipFree(HAT_d);
	hipFree(input_d);
	free(HAT);
	free(input);

	return 0;
}