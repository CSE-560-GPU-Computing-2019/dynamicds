#include "hip/hip_runtime.h"
//Sujay Raj - MT18108
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
typedef struct SlabList SlabList;

#define SLAB_SIZE 4
struct SlabList{
	int val[SLAB_SIZE];
	int key[SLAB_SIZE];
	struct SlabList* next;
};

struct SlabList* createSlablist(struct SlabList* head_ref, int* new_key, int* new_val, int size) { 
	head_ref=NULL;
	for(int i=0;i<size/4;i++){
		struct SlabList* new_node = (struct SlabList*) malloc(sizeof(struct SlabList)); 
		for(int j=0;j<SLAB_SIZE;j++){
    			new_node->key[j] = new_key[i*SLAB_SIZE+j]; 
			new_node->val[j]= new_val[i*SLAB_SIZE+j];
	//		printf("key--->%d\tVal---->%d\n",new_node->key[j],new_node->val[j]);
		} 
    		new_node->next = head_ref; 
     		head_ref    = new_node;
	}
	return head_ref;
} 
void printList(struct SlabList *node) { 
    	while (node != NULL) { 
		for(int i=0;i<SLAB_SIZE;i++){
        		printf("Key: %d\tValue:%d\n",node->key[i],node->val[i]); 		
		}
  	     	node = node->next; 
	} 
} 
void printList1(struct SlabList *node, int size) {
	for(int j=0;j<size;j++) {
		for(int i=0;i<SLAB_SIZE;i++){
        		printf("Key: %d\tValue:%d\n",node[j].key[i],node[j].val[i]); 		
		}
	}
} 
__global__ void insertKernel(struct SlabList* head_ref, int* new_key, int* new_val, int size,struct SlabList* SL, struct SlabList* temp){
//__global__ void insertKernel(struct SlabList* SL){
	printf("INSIDEKERNEL!!!!\n");
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id==0){
//SL->key[0]=1;
//	SL->val[0]=2;
//SL->next=NULL;
	head_ref=NULL;
	for(int i=0;i<size/SLAB_SIZE;i++){
		//temp=NULL;
		struct SlabList* new_node = (struct SlabList*) malloc(sizeof(struct SlabList)); 
		for(int j=0;j<SLAB_SIZE;j++){
    			new_node->key[j] = new_key[i*SLAB_SIZE+j]; 
			new_node->val[j]= new_val[i*SLAB_SIZE+j];
    			//SL->key[j] = new_key[i*SLAB_SIZE+j]; 
			//SL->val[j]= new_val[i*SLAB_SIZE+j];
			printf("key--->%d\tVal---->%d\n",new_node->key[j],new_node->val[j]);
		}
		new_node->next = head_ref; 
    		//SL = SL->next;
		//SL->next = head_ref; 
	memcpy(SL,new_node, size * sizeof(struct SlabList));
     		head_ref    = new_node;
	//	memcpy(temp,new_node, size * sizeof(struct SlabList));
		SL++;
	}
	//SL->next =NULL;
		//SL =SL->next;	
	//return head_ref;
    	/*while (head_ref != NULL) { 
		for(int i=0;i<SLAB_SIZE;i++){
        		printf("Key: %d\tValue:%d\n",head_ref->key[i],head_ref->val[i]); 		
		}
  	     	head_ref = head_ref->next; 
	}*/
//	SL->next =NULL;
	printf("here!!!\n");
		/*while (SL != NULL) { 
		for(int i=0;i<SLAB_SIZE;i++){
        		printf("Key: %d\tValue:%d\n",SL->key[i],SL->val[i]); 		
		}
  	     	SL = SL->next; 
		}*/
}
} 

int main(void){
	int N = 12;
  	int *val_array = (int *)malloc(N * sizeof(int));
  	int *key_array = (int *)malloc(N * sizeof(int));
  	int *d_val_array = NULL;
  	int *d_key_array = NULL;
  	struct SlabList *start=(struct SlabList*)malloc(sizeof(struct SlabList));
  	struct SlabList *d_start=NULL;
	//HashTable = (struct Slab*)malloc(10*sizeof(struct Slab)); 
  	hipMalloc(&d_start, N * sizeof(struct SlabList));
  	hipMalloc(&d_val_array, N * sizeof(int));
  	hipMalloc(&d_key_array, N * sizeof(int));
	for (int i = 0; i < N; i++){
		val_array[i] =i;
		key_array[i] =i+10;   
	}
	const clock_t begin_time = clock();
	//Batch insertion
	start = NULL;
	struct SlabList* head = createSlablist(start, val_array,key_array,N);  	
		//printf("%d\n",head->val[0]);
	float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
        //printf("done initializing\n");
        hipMemcpy(d_val_array, val_array, N * sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(struct SlabList)*N);
        hipMemcpy(d_key_array, key_array, N * sizeof(int), hipMemcpyHostToDevice);
	struct SlabList* d_SL = NULL;
  	hipMalloc(&d_SL, N * sizeof(struct SlabList));
	struct SlabList* d_temp = NULL;
  	hipMalloc(&d_temp, N * sizeof(struct SlabList));	
        const clock_t begin_time1 = clock();
        insertKernel<<<1, 1>>>(d_start,d_val_array,d_key_array,N,d_SL,d_temp);
//        insertKernel<<<1, 1>>>(d_SL);
//       hipDeviceSynchronize();
        struct SlabList* head1 = (struct SlabList*)malloc(N*sizeof(struct SlabList));
        hipMemcpy(head1, d_SL, N * sizeof(struct SlabList), hipMemcpyDeviceToHost);
	//printf("%d\n",head1->key[0]);
        float runTime1 = (float)( clock() - begin_time1 ) /  CLOCKS_PER_SEC;
        printf("Time for matching keywords: %fs\n\n", runTime1);
        printf("____________________________GPU Insertion!!!_______________________________________\n");
        printList1(head1,N/SLAB_SIZE);
	printf("Insert number of elements you want to insert\n");
	int num;
	scanf("%d",&num);
	printf("Enter key and value pair \n");
	for(int i=0;i<num;i++)
		scanf("%d%d",&key_array[i],&val_array[i]);
	printf("Entered key values are\n");
	for(int i=0;i<num;i++)
		printf("Key:%d\tVal:%d\n",key_array[i],val_array[i]);
return 0;
}
