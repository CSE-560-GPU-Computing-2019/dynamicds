#include "hip/hip_runtime.h"
//Krishna Bagaria MT18128
//HAT --- Hashed Array Tree

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define size 1000000   //initial size of HAT

int count_ints (const char* file_name)
{
	FILE* file = fopen (file_name, "r");
	int i = 0;
	int inputsize = 0;
	fscanf (file, "%d", &i);    
	while (!feof (file))
	{  
		//printf ("%d ", i);
		fscanf (file, "%d", &i);
		inputsize++;
	}
	//printf("\n%d",inputsize);
	fclose (file);
	return inputsize;
}

void read_ints (const char* file_name, int * input)
{
	FILE* file = fopen (file_name, "r");
	int i = 0;
	int inputsize = 0;
	fscanf (file, "%d", &i);    
	while (!feof (file))
	{  
		input[inputsize] = i;
		//printf ("%d ", i);
		inputsize++;
		fscanf (file, "%d", &i);
	}
	//printf("%d",inputsize);
	fclose (file);
}

// method to print the HAT
void print_tree(int ** HAT, int n){
	for (int i = 0; i < n; i++){
		if (HAT[i] != NULL){
			printf("\nBucket %d : ",i);
			for (int j =0; j<n ; j++){
				printf("%d ",HAT[i][j]);
			}
		}
		else {
			printf("\nBucket %d is empty.",i);
		}
	}
}

//CPU method to insert elements in HAT from 'input' array passed as argument
void insert_tree (int ** HAT, int n, int * input, int inputsize) {
	for (int i = 0; i < inputsize; i++){
		int j = i/n;
		if (HAT[j]== NULL){
			HAT[j] = (int *)malloc(sizeof(int) * n);
		}
		int k = i % n;
		HAT[j][k] = input[i];
	}
}

//GPU method for inserting elements in HAT
__global__ void insert_gpu (int ** HAT_d, int *input, int inputsize, int n) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	
	if(i < inputsize){
		
		int j = i/n;
		int k = i % n;
		HAT_d[j][k] = input[i];
		
	}
}


int main (int argc, const char **argv) {
	int ** HAT;
	int n = sqrt(size);   //calculate size of main array or each leaf
	HAT = (int **)malloc(sizeof(int *) * n);
	
	int inputsize = count_ints ("data/input10000.txt");
	int * input = (int *)malloc(sizeof(int) * inputsize);
	read_ints("input.txt",input);
	
	/*
	for (int i =0; i<inputsize; i++) {
		printf("%d ", input[i]);
	}
	*/
	
	/*
	printf("Enter no. of elements to be inserted: ");
	scanf("%d",&inputsize);
	
	int * input = (int *)malloc(sizeof(int) * inputsize);
	printf("Enter the elements (integers): ");
	
	for (int i =0; i<inputsize; i++) {
		scanf("%d", &input[i]);
	}
	*/
	
	const clock_t begin_time = clock();  // measure CPU time for insertion
	insert_tree(HAT, n , input, inputsize);
	float runTime_cpu = (float)( clock() - begin_time ) / CLOCKS_PER_SEC;
	
	printf("\nOutput Tree by inserting from CPU:\n");
	print_tree(HAT,n);
	
	
	//GPU code starts here
	int * input_d;
	int ** HAT_d;
	
	hipMalloc ((void **)&input_d , sizeof(int) * inputsize);
	hipMalloc ((void ***)&HAT_d , sizeof(int *) * n);
	
	int **support;
	support = (int**) malloc(sizeof(int*)*n);
	hipMemcpy(support,HAT_d,n*sizeof(int*),hipMemcpyDeviceToHost);
	
	for (int i =0; i<n; i++){
		hipMalloc((void**)&support[i],sizeof(int) * n);
	}
	
	hipMemcpy (input_d, input, sizeof(int) * inputsize , hipMemcpyHostToDevice);
	
	int grid_size = (inputsize % 1024) ? ((inputsize/1024) + 1) : (inputsize/1024);
	int block_size = 1024;
	
	const clock_t begin_time1 = clock(); 
	insert_gpu<<<grid_size,block_size>>>(HAT_d,input_d, inputsize, n);
	//hipDeviceSynchronize();
	float runTime_gpu = (float)( clock() - begin_time1 ) /  CLOCKS_PER_SEC;
	
	
	hipMemcpy (HAT, HAT_d, sizeof(int*) * n , hipMemcpyDeviceToHost);
	
	printf("\nOutput Tree by inserting from GPU:\n");
	print_tree(HAT,n);
	
	printf("Time for inserting(CPU): %fs\n\n", runTime_cpu);
	printf("\n\nTime for inserting(GPU): %fs\n\n", runTime_gpu);
	
	hipFree(HAT_d);
	hipFree(input_d);
	free(HAT);
	free(input);

	return 0;
}