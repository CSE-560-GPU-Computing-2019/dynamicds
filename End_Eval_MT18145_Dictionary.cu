#include "hip/hip_runtime.h"
//MT18145(Shubham Kumar)
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <thrust/sort.h>
#include <algorithm>
#define NUM 16
#define SIZE 16
#define MAXE 10000000
#define BLOCKSIZE 1024
#define INCREMENT 100000
unsigned long int numofelements = 0;
unsigned long int maxelements = MAXE;
unsigned long int size = 0; //Number of estimated key-value pairs in Array of Structure after insertion
unsigned int loop=0;

//key-value pairs are also referred to as elements in this code.
//Structure for dictionary user-defined data type
//Use structure of arrays instead to use sort and merge
//Dictionary is ordered lexicographically and does not search for duplicate
//keys. It supports any number of elements to be inserted
//while executing on GPU. So it is dynamic dictionary on GPU.
//However, batch size during insertion/deletion should be less than maxelements,
//otherwise risizing would become frequent and performance would degrade.
//Key value pairs are stored in lexicographically sorted order.
typedef struct
{
	char key[SIZE];
	char value[SIZE];
	
}Dictionary;

Dictionary *gpu_output_dict;
	//desired Array of Structures on GPU of type Dictionary
	//to store key value pairs on GPU 
float runtime=0.0;

//Kernel to insert elements in existing dynamic GPU dictionary(Array of Structures)
__global__ void insertBatch(Dictionary *dictionary, Dictionary *data, int batchsize, int size)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index<batchsize)
	{
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].key[i] = '\0';
		}	
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].value[i] = '\0';
		}
	}
	__syncthreads();
	if (index<batchsize)
	{
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].key[i] = data[index].key[i];
		}
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].value[i] = data[index].value[i];
		}	
	}
	__syncthreads();	 
}

//Kernel to print elements of the Dictionary between index start and end
__global__ void printDictionary(Dictionary *device_dictionary, int start, int end)
{
	//if (threadIx.x==0)
	for (int i=start;i<=end;i++)
	{
		printf("Key: %s \tValue: %s \n",device_dictionary[i].key,device_dictionary[i].value);
	}
}

//Kernel to copy key-value pairs of two different Dictionary Array of Structures
__global__ void copyongpu(Dictionary *dst, Dictionary *src, int batchsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
		/*if (index<batchsize)
		{
			for (int i=0;i<SIZE;i++) 
			{
				dst[index].key[i]='\0';
				dst[index].value[i]='\0';
			}
		}
		__syncthreads();*/
	if (index<batchsize)
	{
		for (int i=0;i<SIZE;i++) 
		{
			dst[index].key[i] = src[index].key[i];
			dst[index].value[i] = src[index].value[i];	
		}
	
	}
	__syncthreads();

}

//Kernel to search bulk of elements in Dictionary
__global__ void searchElements(Dictionary *temp, int *index_array, char** search_arr, int num, int numofelements)
{
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int flag = -1;
	int tid = threadIdx.x;
	__shared__ char search_arr_shared[1024][SIZE]; 
	if (index<num)
	{
		for (int i=0;i<SIZE;i++)
		search_arr_shared[tid][i]=search_arr[index][i];
	}
	__syncthreads();
	if (index < num)
	{
		for (int i=0;i<numofelements;i++) 
			{
				flag = 0;
				for (int j=0;search_arr_shared[tid][j]!='\0';j++)
				
				{
				
					if (search_arr_shared[tid][j]!=temp[i].key[j])
						{	
							flag = 1;
							break;
						}
					
				}
				if (flag==0)
				{
					index_array[index]=i;
				}

			}
	}
	
	__syncthreads();

}

//Kernel to delete already searched elements by changing the first character of such elements in Dictionary and then eliminating them
//after sorting. Complete logic is in method deleteElements() 
__global__ void deleteElements(Dictionary *temp, int *index_array, int num)
{
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < num)
	{
		int temp_index = index_array[index];

		if (temp_index!=-1)
		{
			temp[temp_index].key[0]='{';
			temp[temp_index].value[0]='$';
		}
	}
	
	__syncthreads();

}

//Comparator function to be passed to sort()
bool compareElements(const Dictionary &gpu_first_dictionary, const Dictionary &gpu_second_dictionary)
{
	int compared_value = 0;
	for (int k=0;k<SIZE;k++)
	{
		if (gpu_first_dictionary.key[k]<gpu_second_dictionary.key[k])
		{
			compared_value  = -1;
			break;
		}
		else if (gpu_first_dictionary.key[k]>gpu_second_dictionary.key[k])
		{
			compared_value = 1;
			break;
		}

	}
	if (compared_value<0)
		return true;
	else
		return false;
};


//Function to handle the logic for insertion of elements into dynamic dictionary on GPU
void insertData()
{	
	char input_string[35];//To take input from user
	int num=NUM;
	printf("Enter number of key-value pairs. Enter 0 to exit. ");
	scanf("%d",&num);
	//If user enters 0 or negative number, program exits
	if (num<0)
	{
		printf("\nInvalid entry by user. Negative value encountered. ");
		return;
	}
	else if (num==0)
	{
		return;
	}
	unsigned int blocksize, numofblocks; //blocksize, numofblocks are used for kernel launch
	hipError_t errors;		
	hipError_t cudaStatus;
	maxelements = MAXE;
	//if size of array of structure on GPU exceeds maxelements,
	//then array of structure is resized,
	//and GPU array of structure is copied to temporary array
	//of structure and again copied back when resizing is done,
	//then insertion is done
	char *token = NULL;//Used to separate key and value inserted by user
	
		if (maxelements<num)
		{
			//batchsize should always be less than number of already allocated elements. Resizing should not be frequent.
			printf("Number of elements to be inserted should be such that resizing is done only after many batches of insertions. Please enter value < 10 million. \n");
			return;
		}
		size+=num;
		//declared as global variable, helps in deletion operation too
		//represents total no of elements after insertion
		//Insertion yet to be done
		
		loop+=1;	
		if (size>maxelements)
		{
			//resizing needed
			int numofblocks2 = 0, blocksize2 = 0;
			//blocksize2, numofblocks2 are used for kernel launch
			while (size>maxelements)
			maxelements += INCREMENT;
			Dictionary *gpu_temp_dict;
			//Array of Structures of type Dictionary
			//to store elements temporarily so that 
			//gpu_output_dict can be resized
			
			errors = hipMalloc((void**)&gpu_temp_dict,maxelements*sizeof(Dictionary));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for gpu_temp_dict: %s\n",hipGetErrorString(errors));
			}
				numofblocks2 = (maxelements%BLOCKSIZE==0)?(maxelements/BLOCKSIZE):(maxelements/BLOCKSIZE+1);
				blocksize2 = BLOCKSIZE;
				printf("Num of blocks %d\n",numofblocks2);
			
			copyongpu<<<numofblocks2,blocksize2>>>((Dictionary*)gpu_temp_dict,(Dictionary*)gpu_output_dict,numofelements);
			
			errors = hipDeviceSynchronize();
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_output_dict to gpu_temp_dict: %s\n",hipGetErrorString(errors));

			}
			
			errors = hipFree(gpu_output_dict);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipFree failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
			}
			errors = hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for resizing gpu_output_dict: %s\n",hipGetErrorString(errors));
			}
			
			copyongpu<<<numofblocks2,blocksize2>>>((Dictionary*)gpu_output_dict,(Dictionary*)gpu_temp_dict,numofelements);
			
			errors = hipDeviceSynchronize();
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_temp_dict to gpu_output_dict copyongpu: %s\n",hipGetErrorString(errors));
			}
			
			errors = hipFree(gpu_temp_dict);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipFree failed for gpu_temp_dict: %s\n",hipGetErrorString(errors));
			}

		}

		clock_t initial_time,finish_time;
		initial_time = clock();
		numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		blocksize = BLOCKSIZE;
		Dictionary *dict = (Dictionary*)malloc(num*sizeof(Dictionary));
		for (int j=0;j<num;j++)
		{
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char *inp2 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(inp2,"Value");
				strcat(inp2,(const char*)jinp);
				strcat(inp2,"_");
				strcat(inp2,cloop);
				strcpy(input_string,inp1);
				strcat(input_string,"-");
				strcat(input_string,inp2);
		
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter key and value separated by -(at most 15 characters each): ");
				scanf("%s",input_string);	
			}
			token = strtok(input_string,"-");
			memset(dict[j].key,'\0',sizeof(dict[j].key));
			memset(dict[j].value,'\0',sizeof(dict[j].value));
			strncpy(dict[j].key,token,SIZE);
			token = strtok(NULL,"-");
			strncpy(dict[j].value,token,SIZE);
			 
		}
		
		Dictionary *gpu_input_dict;
		errors = hipMalloc((void**)&gpu_input_dict,num*sizeof(Dictionary));
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for gpu_input_dict: %s\n",hipGetErrorString(errors));
			}
		errors = hipMemcpy(gpu_input_dict,dict,num*sizeof(Dictionary),hipMemcpyHostToDevice);
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for dict(host) to gpu_input_dict: %s\n",hipGetErrorString(errors));
			}
		insertBatch<<<numofblocks,blocksize>>>(gpu_output_dict,gpu_input_dict,num,numofelements);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus!=hipSuccess)
		{
			fprintf(stderr,"insertBatch kernel failed: %s\n",hipGetErrorString(cudaStatus));
		}
		finish_time = clock();
		runtime = finish_time - initial_time;
		errors = hipFree(gpu_input_dict);
		if(errors!=hipSuccess)
		{
			fprintf(stderr,"hipFree failed for gpu_input_dict: %s\n",hipGetErrorString(errors));
		}
		numofelements+=num;
		free(dict);


		//sort the array of structures
		Dictionary *sort_dict = (Dictionary*)malloc(sizeof(Dictionary)*numofelements);
		errors = hipMemcpy(sort_dict,gpu_output_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for gpu_outut_dict to sort_dict: %s\n",hipGetErrorString(errors));
		}
		std::sort(sort_dict,sort_dict+numofelements,compareElements);
		errors = hipMemcpy(gpu_output_dict,sort_dict,numofelements*sizeof(Dictionary),hipMemcpyHostToDevice);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for sort_dict to gpu_outut_dict: %s\n",hipGetErrorString(errors));
		}
		
		free(sort_dict);
		printf("Time taken for Insertion on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);	
		

}

//Function to handle the logic for search operation applied over dynamic dictionary on GPU
void searchDictionary()
{
	int num=NUM;
	static int loop2=1;
	printf("Enter number of key-value pairs to search for. \n");
	scanf("%d",&num);
	if (num<=0)
	{
		printf("Invalid number. \n");
		return;
	}
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*num);
	for (int i=0;i<num;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
			
	for (int j=0;j<num;j++)
		{
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop2);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(input_string,inp1);
				
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter the key to search for (at most 15 characters in length): ");
				scanf("%s",input_string);	
			}
			
			strncpy(search_array[j],input_string,SIZE);
			
		}
		int *index_array = (int*)malloc(sizeof(int)*num);
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*num);
		char **search_d_array;
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		clock_t initial_time,finish_time;
		initial_time = clock();
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		unsigned int blocksize = 0, numofblocks = 0;
		blocksize = BLOCKSIZE;
		numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		hipMemset(index_d,-1,sizeof(int)*num);
		searchElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		hipDeviceSynchronize();
		finish_time = clock();
		runtime = finish_time - initial_time;
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		
		char keystring[SIZE];
		char valuestring[SIZE];
			
		for (int i=0;i<num;i++)
		{
			memset(keystring,'\0',SIZE*sizeof(char));
			memset(valuestring,'\0',SIZE*sizeof(char));
			if (index_array[i]!=-1)
			{
				hipMemcpy(keystring,gpu_output_dict[index_array[i]].key,sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				hipMemcpy(valuestring,gpu_output_dict[index_array[i]].value,sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				printf("Key: %s, Value: %s\n",keystring,valuestring);
			
			}
			else
			{
				hipMemcpy(keystring,search_h_array[i],sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				printf("Key-Value does not exist for Queried Key: %s\n",keystring);
			}
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);
		printf("Time taken for Lookup on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);
		
		loop2+=1;
}

//Function to handle the logic for delete operation applied over dynamic dictionary on GPU
void deleteDictionary()
{
	static int loop2=1;
	int num=NUM;
	printf("Enter number of key-value pairs to delete from dictionary. \n");
	scanf("%d",&num);
	if (num<=0)
	{
		printf("Invalid number. \n");
		return;
	}
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*num);
	for (int i=0;i<num;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
			
	for (int j=0;j<num;j++)
		{
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop2);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(input_string,inp1);
				
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter the key corrsponding to key-value pair to be deleted(at most 15 characters in length): ");
				scanf("%s",input_string);	
			}
			
			strncpy(search_array[j],input_string,SIZE);
			
		}
		int *index_array = (int*)malloc(sizeof(int)*num);
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*num);
		char **search_d_array;
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		unsigned int blocksize = 0, numofblocks = 0;
		blocksize = BLOCKSIZE;
		numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		hipMemset(index_d,-1,sizeof(int)*num);
		clock_t initial_time,finish_time;
		initial_time = clock();
		searchElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		//Found elements would be deleted. Two step for deletion - search then if found, delete
		hipDeviceSynchronize();
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		int count = 0;
		for (int loop=0;loop<num;loop++)
		{
			if (index_array[loop]!=-1)
				count++;
		}

		hipError_t errors;
		deleteElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,num);
		hipDeviceSynchronize();
		finish_time = clock();
		runtime = finish_time - initial_time;
		
		for (int i=0;i<num;i++)
		{
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);

		Dictionary *sort_dict = (Dictionary*)malloc(sizeof(Dictionary)*numofelements);
		errors = hipMemcpy(sort_dict,gpu_output_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for gpu_outut_dict to sort_dict: %s\n",hipGetErrorString(errors));
		}
		std::sort(sort_dict,sort_dict+numofelements,compareElements);
		errors = hipMemcpy(gpu_output_dict,sort_dict,numofelements*sizeof(Dictionary),hipMemcpyHostToDevice);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for sort_dict to gpu_outut_dict: %s\n",hipGetErrorString(errors));
		}
		
		free(sort_dict);

		printf("Found key-value pairs Deleted. \n");
		numofelements = numofelements - count;
		size = size - count;

		loop2+=1;
		printf("Time taken for Deletion on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);

}

//Function to handle the logic for Range Query Operation(Count+Lookup) applied over dynamic dictionary on GPU
void rangeQuery()
{
	int num=NUM;
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*2);
	for (int i=0;i<2;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
	num = 2; //num is 2 as only two keys are provided by user
	for (int j=0;j<2;j++)
		{
			printf("Enter the key(Key must be exisitng in dictionary): ");
			scanf("%s",input_string);	
			
			strncpy(search_array[j],input_string,SIZE);
			
		}

		int *index_array = (int*)malloc(sizeof(int)*2);
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*2);
		char **search_d_array;
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		clock_t initial_time,finish_time;
		initial_time = clock();
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		hipMemset(index_d,-1,sizeof(int)*num);
		searchElements<<<1,2>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		hipDeviceSynchronize();
		finish_time = clock();
		runtime = finish_time - initial_time;
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		
		
		int count = 0;
		if (index_array[0]!=-1)
			count++;
		if (index_array[1]!=-1)
			count++;
		
		if (count!=2)
		{
			printf("One or more key not found in dictionary. Please enter existing key. \n");
			return;
		}
		if (index_array[1]<index_array[0])
			{
				int temp_var = index_array[0];
				index_array[0] = index_array[1];
				index_array[1] = temp_var;
			}
		int rangecount = index_array[1]-index_array[0]+1;

		printf("Number of elements between elements corresponding to given two keys are: %d\n",rangecount);
		printf("Elements are: \n");
		printDictionary<<<1,1>>>(gpu_output_dict,index_array[0],index_array[1]);
		hipDeviceSynchronize();

		for (int i=0;i<num;i++)
		{
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);
		printf("Time taken for Range Query on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);
		

}

int main()
{
	hipError_t errors;
	
	char user_input[50];
	memset(user_input,'\0',50);
	errors = hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
	//float runtime=0.0;
	//clock_t initial_time,finish_time;
	if(errors!=hipSuccess)
	{
		fprintf(stderr,"hipMalloc failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
		return 0;
	}
	
	do
	{
		printf("\n\n**********************************************CHOICES**********************************************\n\n");
		printf("Enter insert for Insert opeation in batches. \n");
		printf("Enter delete for Delete operation in batches. \n");
		printf("Enter showlast for viewing last 10 elements in the dictionary. \n");
		printf("Enter showfirst for viewing first 10 elements in the dictionary. \n");
		printf("Enter search for search/lookup operation over all elements. \n");
		printf("Enter range-search for getting list of key-value pairs and count between a particular range of existing keys. \n");
		printf("\n\n***************************************************************************************************\n\n");
		printf("Enter your choice [insert/delete/showlast/showfirst/search/range-search/exit]: ");
		scanf("%s",user_input);
			
		if (strncmp(user_input,"insert",6)==0)
		{
			printf("Insert operation. \n");
			//initial_time = clock();
			insertData();
			//finish_time = clock();
			//runtime = finish_time - initial_time;
			//printf("Time taken for Insertion on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);
		}
		else if (strncmp(user_input,"delete",6)==0)
		{
			if (numofelements==0)
			{
				printf("Deletion not possible. Dictionary is empty. \n");
			}
			else
			{
				printf("Delete operation. \n");
				//initial_time = clock();
				deleteDictionary();
				//finish_time = clock();
				//runtime = finish_time - initial_time;
				//printf("Time taken for Deletion on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);		
			}
			
		}
		else if (strncmp(user_input,"showlast",8)==0)
		{
			if (numofelements==0)
			{
				printf("Can't display last 10 elements. Dictionary is empty. \n");
			}
			else if (numofelements > 0)
			{
				printf("\n-------------------------------------------------\n");
				printf("Last 10 elements of Dictionary: \n");
				int j=(size>10)?(size-10):0;
				
				printDictionary<<<1,1>>>(gpu_output_dict,j,size-1);
				errors = hipDeviceSynchronize();
				if (errors!=hipSuccess)
				{
					printf("Error in printing first 10 elements of array. ");
				}
				printf("\n-------------------------------------------------\n");
				
			}
			
		}
		else if (strncmp(user_input,"showfirst",9)==0)
		{
			if (numofelements==0)
			{
				printf("Can't display first 10 elements. Dictionary is empty. \n");
			}
			else if (numofelements > 0)
			{
				printf("\n-------------------------------------------------\n");
				printf("First 10 elements of Dictionary: \n");
				int start = 0;
				int end = (size<10)?size-1:9;
				printDictionary<<<1,1>>>(gpu_output_dict,start,end);
				errors = hipDeviceSynchronize();
				if (errors!=hipSuccess)
				{
					printf("Error in printing first 10 elements of array. ");
				}
				printf("\n-------------------------------------------------\n");
		
				}
			}
		else if (strncmp(user_input,"search",6)==0)
		{
			if (numofelements==0)
			{
				printf("Can't do search operation. Dictionary is empty. \n");
			}
			else
			{
				printf("Search operation. \n");
				//initial_time = clock();		
				searchDictionary();
				//finish_time = clock();
				//runtime = finish_time - initial_time;
				//printf("Time taken for Lookup operation on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);
			}
			
		}
		else if (strncmp(user_input,"range-search",12)==0)
		{
			if (numofelements==0)
			{
				printf("Can't do range-search operation. Dictionary is empty. \n");
			}
			else
			{
				printf("Range-Search operation. \n");
				//initial_time = clock();		
				rangeQuery();
				//finish_time = clock();
				//runtime = finish_time - initial_time;
				//printf("Time taken for Range search/Count operation on GPU: %fs.\n",(float)runtime/CLOCKS_PER_SEC);
				
			}
			
		}
		else if (strncmp(user_input,"exit",4)!=0)
		{
			printf("Invalid input. Enter valid input. \n");
			
		}	 

	
	}while(strncmp(user_input,"exit",4)!=0);
	
	errors = hipFree(gpu_output_dict);
	
	if(errors!=hipSuccess)
	{
		fprintf(stderr,"hipFree failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
	}

	
	return 0;
	
}
