#include "hip/hip_runtime.h"
//MT18145(Shubham Kumar)
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <thrust/sort.h>
#include <algorithm>
#define NUM 16
#define SIZE 16
#define MAXE 10000000
#define BLOCKSIZE 1024
#define INCREMENT 100000
unsigned long int numofelements = 0;
unsigned long int maxelements = MAXE;
unsigned long int size = 0; //Number of estimated key-value pairs in Array of Structure after insertion
unsigned int loop=0;
__device__ unsigned int found_flag = 0;//flag to be used to make other threads do the minimal work
//key-value pairs are also referred to as elements in this code.
//Structure for dictionary user-defined data type
//Use structure of arrays instead to use sort and merge
//Dictionary is not ordered and does not search for duplicate
//keys. It supports any number of elements to be inserted
//while executing on GPU. So it is dynamic dictionary.
//Key value pairs are stored in order in which they are inserted.
typedef struct
{
	char key[SIZE];
	char value[SIZE];
	//char* key;
	//char* value;
}Dictionary;

Dictionary *gpu_output_dict;
	//desired Array of Structures on GPU of type Dictionary
	//to store key value pairs on GPU 
	//with inclusion of new key value pairs recently inserted
	


//batchsize < size(in this project for other operations, not insertion)
//Kernel to insert elements in existing dynamic GPU dictionary(Array of Structures)
__global__ void insertBatch(Dictionary *dictionary, Dictionary *data, int batchsize, int size)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index<batchsize)
	{
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].key[i] = '\0';
			//dictionary[index+size].key[i] = data[index].key[i];
		}
		for (int i=0;i<SIZE;i++)
		{
			dictionary[index+size].value[i] = '\0';
			//dictionary[index+size].value[i] = data[index].value[i];
		}
	}
	__syncthreads();
	if (index<batchsize)
	{
		//atomicAdd(count,1);
		for (int i=0;i<SIZE;i++)
		{
			//dictionary[index+size].key[i] = '\0';
			dictionary[index+size].key[i] = data[index].key[i];
		}
		//for (int i=0;i<value_len;i++)
		for (int i=0;i<SIZE;i++)
		{
			//dictionary[index+size].value[i] = '\0';
			dictionary[index+size].value[i] = data[index].value[i];
		}	
	}
	__syncthreads();
	
	 
}
__global__ void printDictionary(Dictionary *device_dictionary, int start, int end)
{
	//if (threadIx.x==0)
	for (int i=start;i<=end;i++)
	{
		printf("Key: %s \tValue: %s \n",device_dictionary[i].key,device_dictionary[i].value);
	}
}
//Kernel to copy key-value pairs of two different Dictionary Array of Structures
//__global__ void copyongpu(Dictionary** dst,Dictionary** src, int batchsize)
__global__ void copyongpu(Dictionary *dst, Dictionary *src, int batchsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//if (index==0)
	//		*count=0;
	//	__syncthreads();
		if (index<batchsize)
		{
			for (int i=0;i<SIZE;i++) 
			{
				dst[index].key[i]='\0';
				dst[index].value[i]='\0';
				//dst[index].key[i] = src[index].key[i];
				//dst[index].value[i] = src[index].value[i];	
			}
		}
		__syncthreads();
	if (index<batchsize)
	{
		//atomicAdd(count,1);
		for (int i=0;i<SIZE;i++) 
		{
			//dst[index].key[i]='\0';
			//dst[index].value[i]='\0';
			dst[index].key[i] = src[index].key[i];
			dst[index].value[i] = src[index].value[i];	
		}
	
	}
	__syncthreads();
		
	
	//__syncthreads();
	//if (index>100000)
	//	printf("Index: %d\n", index);
}
__global__ void searchElements(Dictionary *temp, int *index_array, char** search_arr, int num, int numofelements)
{
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int flag = -1;
	
	__syncthreads();
	if (index < num)
	{
		for (int i=0;i<numofelements;i++) 
			{
				flag = 0;
				for (int j=0;search_arr[index][j]!='\0';j++)
				{
					//printf("Search arr: %c\n",search_arr[index][j]);
					//printf("Present element: %c\n",temp[i].key[j]);

					if (search_arr[index][j]!=temp[i].key[j])
						{	
							//int temp = flag+1;
							flag = 1;
							break;
						}
					
				}
				if (flag==0)
				{
					//printf("Index: %d\n",i);
					index_array[index]=i;
					//return;
					//return;
				}

			}
	}
	
	__syncthreads();

}

__global__ void deleteElements(Dictionary *temp, int *index_array, int num)
{
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < num)
	{
		int temp_index = index_array[index];

		if (temp_index!=-1)
		{
			temp[temp_index].key[0]='{';
			temp[temp_index].value[0]='$';
		}
	}
	
	__syncthreads();

}


/*
__host__ __device__ 
*/
bool compareElements(const Dictionary &gpu_first_dictionary, const Dictionary &gpu_second_dictionary)
{
	//if (strncmp(*gpu_first_dictionary.value,*gpu_second_dictionary.value)!=0)
	//int comparebits = 15;
	int compared_value = 0;
	for (int k=0;k<SIZE;k++)
	{
		if (gpu_first_dictionary.key[k]<gpu_second_dictionary.key[k])
		{
			compared_value  = -1;
			break;
		}
		else if (gpu_first_dictionary.key[k]>gpu_second_dictionary.key[k])
		{
			compared_value = 1;
			break;
		}

	}
	//strncmp(gpu_first_dictionary.key,gpu_second_dictionary.key,comparebits);
	if (compared_value<0)
		return true;
	else
		return false;
	//else
		//return strncmp(*gpu_first_dictionary.value,*gpu_second_dictionary.key);
};
/**/


void insertData()
{	
	char input_string[35];//To take input from user
	int num=NUM;
	//int size = 0;
	//int numofelements = 0;// Number of existing key-value pairs
	printf("Enter number of key-value pairs. Enter 0 to exit. ");
	scanf("%d",&num);
	//If user enters 0 or negative number, program exits
	if (num<0)
	{
		printf("\nInvalid entry by user. Negative value encountered. ");
		return;
	}
	else if (num==0)
	{
		return;
	}
	unsigned int blocksize, numofblocks; //blocksize, numofblocks are used for kernel launch
	hipError_t errors;		
	hipError_t cudaStatus;
	maxelements = MAXE;
	//if size of array of structure on GPU exceeds maxelements,
	//then array of structure is resized,i.e., maxelements+=1000000
	//and GPU array of structure is copied to temporary array
	//of structure and again copied back when resizing is done,
	//then insertion is done
	char *token = NULL;//Used to separate key and value inserted by user
	//Iteration starts
	//User would enter key-value pairs atleast once if num>0 from
	//input taken above
	//Iteration ends when user enters 0 or a negative number
	
	//do
	//{
		if (maxelements<num)
		{
			//batchsize should always be less than number of existing elements
			printf("Number of elements to be inserted should be such that resizing is done only after many batches of insertions. Exiting. \n");
			return;
		}
		size+=num;
		//represents total no of elements after insertion
		//Insertion yet to be done
		//printf("Number of elements: %d\n",numofelements);
		//printf("Number of elements estimated after insertion: %d\n",size);
		//const clock_t begin_time2 = clock();
		
		loop+=1;	
		if (size>maxelements)
		{//resizing needed
			int numofblocks2 = 0, blocksize2 = 0;
			//blocksize2, numofblocks2 are used for kernel launch
			while (size>maxelements)
			maxelements += INCREMENT;
			Dictionary *gpu_temp_dict;
			//Array of Structures of type Dictionary
			//to store elements temporarily so that 
			//gpu_output_dict can be resized
			
			errors = hipMalloc((void**)&gpu_temp_dict,maxelements*sizeof(Dictionary));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for gpu_temp_dict: %s\n",hipGetErrorString(errors));
			}
			/*for (int j=0;j<maxelements;j++)
			{
			//limit on word length of key and value <= 100 characters
				hipMemset(gpu_temp_dict[j].key,'\0',sizeof(gpu_output_dict[j].key));
				hipMemset(gpu_temp_dict[j].value,'\0',sizeof(gpu_output_dict[j].value));
			 
			}*/
			//printf("Maxelements :%d\n",maxelements);	
			//if (maxelements>1024)
			//{
				numofblocks2 = (maxelements%BLOCKSIZE==0)?(maxelements/BLOCKSIZE):(maxelements/BLOCKSIZE+1);
				blocksize2 = BLOCKSIZE;
				printf("Num of blocks %d\n",numofblocks2);
			//}
			/*else
			{
				numofblocks2 = 1;
				blocksize2 = maxelements;	
			}*/
			//errors = hipMemcpy(gpu_temp_dict,gpu_output_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToDevice);
			//errors = hipMemcpy(cpu_output_dict,gpu_output_dict,maxelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
			/*if (errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for gpu_output_dict to gpu_temp_dict: %s\n",hipGetErrorString(errors));
			}*/

			copyongpu<<<numofblocks2,blocksize2>>>((Dictionary*)gpu_temp_dict,(Dictionary*)gpu_output_dict,numofelements);
			/*cudaStatus = hipGetLastError();
			if (cudaStatus!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_output_dict to gpu_temp_dict: %s\n",hipGetErrorString(cudaStatus));
			}*/

			errors = hipDeviceSynchronize();
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_output_dict to gpu_temp_dict: %s\n",hipGetErrorString(errors));

				//fprintf(stderr,"hipDeviceSynchronize failed for gpu_output_dict to gpu_temp_dict copyongpu: %s\n",hipGetErrorString(errors));
			}
			
			//*cnt_host2 = 0;
			/*errors = hipMemcpy(cnt_host2,cnt2,sizeof(int),hipMemcpyDeviceToHost);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for cnt2 to cnt_host2: %s\n",hipGetErrorString(errors));
			}
			printf("Count from copyongpu kernel: %d\n",*cnt_host2);
			errors = hipMemset(cnt2,0,sizeof(int));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemset failed for count2: %s\n",hipGetErrorString(errors));
			}*/
			errors = hipFree(gpu_output_dict);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipFree failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
			}
			errors = hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for resizing gpu_output_dict: %s\n",hipGetErrorString(errors));
			}
			/*for (int j=0;j<maxelements;j++)
			{
			//limit on word length of key and value <= 100 characters
				hipMemset(gpu_output_dict[j].key,'\0',sizeof(gpu_output_dict[j].key));
				hipMemset(gpu_output_dict[j].value,'\0',sizeof(gpu_output_dict[j].value));
			 
			}*/
			copyongpu<<<numofblocks2,blocksize2>>>((Dictionary*)gpu_output_dict,(Dictionary*)gpu_temp_dict,numofelements);
			/*cudaStatus = hipGetLastError();
			if (cudaStatus!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_temp_dict to gpu_output_dict: %s\n",hipGetErrorString(cudaStatus));
			}*/
			errors = hipDeviceSynchronize();
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"copyongpu kernel failed for gpu_temp_dict to gpu_output_dict copyongpu: %s\n",hipGetErrorString(errors));
			}
			//errors = hipMemcpy(gpu_output_dict,gpu_temp_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToDevice);
			/*if (errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for gpu_temp_dict to gpu_output_dict: %s\n",hipGetErrorString(errors));
			}*/
			/*errors = hipMemcpy(cnt_host2,cnt2,sizeof(int),hipMemcpyDeviceToHost);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for cnt2 to cnt_host2: %s\n",hipGetErrorString(errors));
			}
			printf("Count from copyongpu kernel: %d\n",*cnt_host2);
			errors = hipMemset(cnt2,0,sizeof(int));
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemset failed for count2: %s\n",hipGetErrorString(errors));
			}*/
			errors = hipFree(gpu_temp_dict);
			if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipFree failed for gpu_temp_dict: %s\n",hipGetErrorString(errors));
			}

		}
		
		//if (num>1024)
		//{
			numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
			blocksize = BLOCKSIZE;
		//}
		/*else
		{
			numofblocks = 1;
			blocksize = num;	
		}*/
		//char inpt = '1';
		//char jinp = '1';
		Dictionary *dict = (Dictionary*)malloc(num*sizeof(Dictionary));
		//Dictionary dict[num];
		for (int j=0;j<num;j++)
		{//limit on word length of key and value <= 100 characters
		//	printf("Enter key and value separated by - : ");
			
		//	scanf("%s",input_string);
			//printf("Input: %s",input_string);
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char *inp2 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(inp2,"Value");
				strcat(inp2,(const char*)jinp);
				strcat(inp2,"_");
				strcat(inp2,cloop);
				strcpy(input_string,inp1);
				strcat(input_string,"-");
				strcat(input_string,inp2);
		
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter key and value separated by -(at most 15 characters each): ");
				scanf("%s",input_string);	
			}
			token = strtok(input_string,"-");
			memset(dict[j].key,'\0',sizeof(dict[j].key));
			memset(dict[j].value,'\0',sizeof(dict[j].value));
			strncpy(dict[j].key,token,SIZE);
			//printf("Key: %s\n",dict[j].key);
			token = strtok(NULL,"-");
			strncpy(dict[j].value,token,SIZE);
			//printf("Value: %s\n",dict[j].value);
			 
			//dict[j].key = (char*)malloc(100);
			//dict[j].value = (char*)malloc(100);
			//jinp=jinp+1;
			//printf("Enter value: ");
			//inpt=inpt+'1';
		}
		Dictionary *gpu_input_dict;
		errors = hipMalloc((void**)&gpu_input_dict,num*sizeof(Dictionary));
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMalloc failed for gpu_input_dict: %s\n",hipGetErrorString(errors));
			}
		errors = hipMemcpy(gpu_input_dict,dict,num*sizeof(Dictionary),hipMemcpyHostToDevice);
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for dict(host) to gpu_input_dict: %s\n",hipGetErrorString(errors));
			}
		//const clock_t begin_time = clock();
		//hipMemset((void**)&gpu_output_dict,'\0',num*sizeof(Dictionary));
		insertBatch<<<numofblocks,blocksize>>>(gpu_output_dict,gpu_input_dict,num,numofelements);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus!=hipSuccess)
		{
			fprintf(stderr,"insertBatch kernel failed: %s\n",hipGetErrorString(cudaStatus));
		}
		/*errors = hipMemcpy(cnt_host1,cnt1,sizeof(int),hipMemcpyDeviceToHost);
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for cnt1 to cnt_host1: %s\n",hipGetErrorString(errors));
			}*/
		//printf("Count from insertBatch kernel: %d\n",*cnt_host1);
		/*errors = hipMemset(cnt1,0,sizeof(int));
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemset failed for count1: %s\n",hipGetErrorString(errors));
			}*/
		//errors = hipDeviceSynchronize();
		//float runtime_insert = (float)(clock()-begin_time)/CLOCKS_PER_ ;
		//printf("Insertion time(Only Kernel launch) for %d elements on GPU: %fsec\n",num,runtime_insert);

		//Dictionary *cpu_output_dict = (Dictionary*)malloc(maxelements*sizeof(Dictionary));
		//memset(cpu_output_dict,'\0',sizeof(Dictionary)*num);
		/*Dictionary cpu_output_dict[maxelements];
		for (int j=0;j<maxelements;j++)
		{//limit on word length of key and value <= 100 characters
			memset(cpu_output_dict[j].key,'\0',sizeof(cpu_output_dict[j].key));
			memset(cpu_output_dict[j].value,'\0',sizeof(cpu_output_dict[j].value));
			 
		}
		*/
		/*errors = hipMemcpy(cpu_output_dict,gpu_output_dict,maxelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipMemcpy failed for gpu_output_dict to cpu_output_dict: %s\n",hipGetErrorString(errors));
			}
			*/
		errors = hipDeviceSynchronize();
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipDeviceSynchronize failed for insertBatch: %s\n",hipGetErrorString(errors));
			}
		//float runtime_insert2 = (float)(clock()-begin_time2)/CLOCKS_PER_SEC;
		//printf("Insertion time(Kernel launch+Memory calls) for %d elements on GPU: %fsec\n",num,runtime_insert2);

		//free(dict);
		//free(cpu_output_dict);
		errors = hipFree(gpu_input_dict);
		if(errors!=hipSuccess)
			{
				fprintf(stderr,"hipFree failed for gpu_input_dict: %s\n",hipGetErrorString(errors));
			}
		numofelements+=num;
		//printf("Enter number of key-value pairs. Enter 0 to exit. ");
		//scanf("%d",&num);
		//free(cpu_output_dict);
		free(dict);


		//sort the array of structures
		Dictionary *sort_dict = (Dictionary*)malloc(sizeof(Dictionary)*numofelements);
		errors = hipMemcpy(sort_dict,gpu_output_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for gpu_outut_dict to sort_dict: %s\n",hipGetErrorString(errors));
		}
		std::sort(sort_dict,sort_dict+numofelements,compareElements);
		errors = hipMemcpy(gpu_output_dict,sort_dict,numofelements*sizeof(Dictionary),hipMemcpyHostToDevice);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for sort_dict to gpu_outut_dict: %s\n",hipGetErrorString(errors));
		}
		
		free(sort_dict);

	//}while (num>0);

	/*
	errors = hipFree(gpu_output_dict);
	
	if(errors!=hipSuccess)
	{
		fprintf(stderr,"hipFree failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
	}
	*/

}
void searchDictionary()
{
	int num=NUM;
	static int loop2=1;
	printf("Enter number of key-value pairs to search for. \n");
	scanf("%d",&num);
	if (num<=0)
	{
		printf("Invalid number. \n");
		return;
	}
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*num);
	for (int i=0;i<num;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
	//memset(search_array,'\0',sizeof(char)*num*SIZE);
			
	for (int j=0;j<num;j++)
		{
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop2);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(input_string,inp1);
				//printf("Key to search for: %s .\n",input_string);
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter the key to search for (at most 15 characters in length): ");
				scanf("%s",input_string);	
			}
			
			strncpy(search_array[j],input_string,SIZE);
			//printf("Search array: %s. \n",search_array[j]);
		}
		int *index_array = (int*)malloc(sizeof(int)*num);
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*num);
		char **search_d_array;
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		//for (int i=0;i<num;i++)
		//	printf("Elements: %s\n",search_h_array[i]);
		unsigned int blocksize = 0, numofblocks = 0;
		blocksize = BLOCKSIZE;
		numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		//locksize = 1, numofblocks = 1;
		//for (int element = 0;element<num;element++)
		hipMemset(index_d,-1,sizeof(int)*num);
		searchElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		//errors = 
		hipDeviceSynchronize();
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		/*if (errors!=hipSuccess)
		{
			printf("Error in hipMemcpy from index host to index device. \n");
		}*/

		char keystring[SIZE];
		char valuestring[SIZE];
			
		for (int i=0;i<num;i++)
		{
			memset(keystring,'\0',SIZE*sizeof(char));
			memset(valuestring,'\0',SIZE*sizeof(char));
			//hipMemcpy(keystring,search_h_array[i],sizeof(char)*SIZE,hipMemcpyDeviceToHost);
			//printf("Indices for key %s is : %d.\n",keystring,index_array[i]);
			if (index_array[i]!=-1)
			{
				hipMemcpy(keystring,gpu_output_dict[index_array[i]].key,sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				hipMemcpy(valuestring,gpu_output_dict[index_array[i]].value,sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				printf("Key: %s, Value: %s\n",keystring,valuestring);
			
			}
			else
			{
				hipMemcpy(keystring,search_h_array[i],sizeof(char)*SIZE,hipMemcpyDeviceToHost);
				printf("Key-Value does not exist for Queried Key: %s\n",keystring);
			}
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);
		loop2+=1;
}

void deleteDictionary()
{
	static int loop2=1;
	int num=NUM;
	printf("Enter number of key-value pairs to delete from dictionary. \n");
	scanf("%d",&num);
	if (num<=0)
	{
		printf("Invalid number. \n");
		return;
	}
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*num);
	for (int i=0;i<num;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
	//memset(search_array,'\0',sizeof(char)*num*SIZE);
			
	for (int j=0;j<num;j++)
		{
			char *inp1 = (char*)malloc(SIZE*sizeof(char));
			char jinp[10];
			char cloop[5];
			if (num>5)
			{
				sprintf(cloop,"%d",loop2);
				sprintf(jinp,"%d",j);
				strcpy(inp1,"Key");
				strcat(inp1,(const char*)jinp);
				strcat(inp1,"_");
				strcat(inp1,cloop);
				strcpy(input_string,inp1);
				//printf("Key to search for: %s .\n",input_string);
			}
			
			else if (num>0 && num<=5)
			{
				printf("Enter the key corrsponding to key-value pair to be deleted(at most 15 characters in length): ");
				scanf("%s",input_string);	
			}
			
			strncpy(search_array[j],input_string,SIZE);
			//printf("Search array: %s. \n",search_array[j]);
		}
		int *index_array = (int*)malloc(sizeof(int)*num);
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*num);
		char **search_d_array;
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		//for (int i=0;i<num;i++)
		//	printf("Elements: %s\n",search_h_array[i]);
		unsigned int blocksize = 0, numofblocks = 0;
		blocksize = BLOCKSIZE;
		numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		//locksize = 1, numofblocks = 1;
		//for (int element = 0;element<num;element++)
		hipMemset(index_d,-1,sizeof(int)*num);
		searchElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		//errors = 
		hipDeviceSynchronize();
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		/*if (errors!=hipSuccess)
		{
			printf("Error in hipMemcpy from index host to index device. \n");
		}*/
		int count = 0;
		for (int loop=0;loop<num;loop++)
		{
			if (index_array[loop]!=-1)
				count++;
		}

		//char keystring[SIZE];
		//char valuestring[SIZE];
		hipError_t errors;
		deleteElements<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,num);
		hipDeviceSynchronize();

		for (int i=0;i<num;i++)
		{
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);

		Dictionary *sort_dict = (Dictionary*)malloc(sizeof(Dictionary)*numofelements);
		errors = hipMemcpy(sort_dict,gpu_output_dict,numofelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for gpu_outut_dict to sort_dict: %s\n",hipGetErrorString(errors));
		}
		std::sort(sort_dict,sort_dict+numofelements,compareElements);
		errors = hipMemcpy(gpu_output_dict,sort_dict,numofelements*sizeof(Dictionary),hipMemcpyHostToDevice);
		if (errors!=hipSuccess)
		{
			fprintf(stderr,"hipMemcpy failed for sort_dict to gpu_outut_dict: %s\n",hipGetErrorString(errors));
		}
		
		free(sort_dict);

		printf("Found key-value pairs Deleted. \n");
		numofelements = numofelements - count;
		size = size - count;

		loop2+=1;
}

void rangeQuery()
{
	//static int loop2=1;
	int num=NUM;
	//printf("Enter number of key-value pairs to delete from dictionary. \n");
	//scanf("%d",&num);
	char input_string[SIZE];
	char **search_array = (char**)malloc(sizeof(char*)*2);
	//char search_array[2][SIZE];
	for (int i=0;i<2;i++)
	{
		search_array[i] = (char*)malloc	(sizeof(char)*SIZE);
		memset(search_array[i],'\0',sizeof(char)*SIZE);
	}
			num = 2;
	for (int j=0;j<2;j++)
		{
			printf("Enter the key(Key must be exisitng in dictionary): ");
			scanf("%s",input_string);	
			
			strncpy(search_array[j],input_string,SIZE);
			//printf("Search array: %s. \n",search_array[j]);
		}

		int *index_array = (int*)malloc(sizeof(int)*2);
		//int index_array[2];
		//TO store indices of elements
		int *index_d;
		hipMalloc((void**)&index_d,sizeof(int)*2);
		char **search_d_array;
		//char search_d_array
		char **search_h_array = (char**)malloc(sizeof(char*)*num);
		hipMalloc((void**)&search_d_array,sizeof(char*)*num);
		hipMemcpy(search_h_array,search_d_array,sizeof(char*)*num,hipMemcpyDeviceToHost);
		for (int i=0;i<num;i++)
		{
			hipMalloc((void**)&search_h_array[i],sizeof(char)*SIZE);
			hipMemset(search_h_array[i],'\0',sizeof(char)*SIZE);
			hipMemcpy(search_h_array[i],search_array[i],sizeof(char)*SIZE,hipMemcpyHostToDevice);
		}
		hipMemcpy(search_d_array,search_h_array,sizeof(char*)*num,hipMemcpyHostToDevice);
		//for (int i=0;i<num;i++)
		//	printf("Elements: %s\n",search_h_array[i]);
		unsigned int blocksize = 0, numofblocks = 0;
		//blocksize = BLOCKSIZE;
		//numofblocks = (num%BLOCKSIZE==0)?(num/BLOCKSIZE):(num/BLOCKSIZE+1);
		//locksize = 1, numofblocks = 1;
		//for (int element = 0;element<num;element++)
		hipMemset(index_d,-1,sizeof(int)*num);
		searchElements<<<1,2>>>(gpu_output_dict,index_d,search_d_array,num,numofelements);
		//errors = 
		hipDeviceSynchronize();
		hipMemcpy(index_array,index_d,sizeof(int)*num,hipMemcpyDeviceToHost);
		/*if (errors!=hipSuccess)
		{
			printf("Error in hipMemcpy from index host to index device. \n");
		}*/
		int count = 0;
		if (index_array[0]!=-1)
			count++;
		if (index_array[1]!=-1)
			count++;
		
		if (count!=2)
		{
			printf("One or more key not found in dictionary. Please enter existing key. \n");
			return;
		}
		if (index_array[1]<index_array[0])
			{
				int temp_var = index_array[0];
				index_array[0] = index_array[1];
				index_array[1] = temp_var;
			}
		int rangecount = index_array[1]-index_array[0]+1;

		printf("Number of elements between elements corresponding to given two keys are: %d\n",rangecount);
		printf("Elements are: \n");
		printDictionary<<<1,1>>>(gpu_output_dict,index_array[0],index_array[1]);
		//char keystring[SIZE];
		//char valuestring[SIZE];
		//hipError_t errors;
		//printDictionary<<<numofblocks,blocksize>>>(gpu_output_dict,index_d,num);
		hipDeviceSynchronize();

		for (int i=0;i<num;i++)
		{
			hipFree(search_h_array[i]);

		}
		for (int i=0;i<num;i++)
			free(search_array[i]);
		free(search_array);
		free(index_array);
		hipFree(index_d);
		hipFree(search_d_array);

		
		//loop2+=1;
}

int main()
{
	//int num = NUM;
	//num represents number of elements user wants to enter
	//in any iteration
	hipError_t errors;
	
	char user_input[50];
	memset(user_input,'\0',50);
	errors = hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
	if(errors!=hipSuccess)
	{
		fprintf(stderr,"hipMalloc failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
	}
	
	do
	{
		printf("\n\n**********************************************CHOICES**********************************************\n\n");
		printf("Enter insert for Insert opeation in batches. \n");
		printf("Enter delete for Delete operation in batches. \n");
		printf("Enter showlast for viewing last 10 elements in the dictionary. \n");
		printf("Enter showfirst for viewing first 10 elements in the dictionary. \n");
		printf("Enter search for search/lookup operation over all elements. \n");
		printf("Enter range-search for getting list of key-value pairs and count between a particular range of existing keys. \n");
		printf("\n\n***************************************************************************************************\n\n");
		printf("Enter your choice [insert/delete/showlast/showfirst/search/range-search/exit]: ");
		scanf("%s",user_input);
		hipEvent_t start,stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
			
		if (strncmp(user_input,"insert",6)==0)
		{
			printf("Insert operation. \n");
			hipEventRecord(start,0);
			insertData();
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time,start,stop);
			printf("Time taken for Insertion on GPU: %fms.\n",time/1000);
		}
		else if (strncmp(user_input,"delete",6)==0)
		{
			if (numofelements==0)
			{
				printf("Deletion not possible. Dictionary is empty. \n");
			}
			else
			{
				printf("Delete operation. \n");
				hipEventRecord(start,0);
				deleteDictionary();
				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time,start,stop);
				printf("Time taken for Deletion on GPU: %fms.\n",time/1000);		
			}
			
		}
		else if (strncmp(user_input,"showlast",8)==0)
		{
			if (numofelements==0)
			{
				printf("Can't display last 10 elements. Dictionary is empty. \n");
			}
			else if (numofelements > 0)
			{
				printf("\n-------------------------------------------------\n");
				printf("Last 10 elements of Dictionary: \n");
				int j=(size>10)?(size-10):0;
				
				printDictionary<<<1,1>>>(gpu_output_dict,j,size-1);
				errors = hipDeviceSynchronize();
				if (errors!=hipSuccess)
				{
					printf("Error in printing first 10 elements of array. ");
				}
				printf("\n-------------------------------------------------\n");
				
			}
			
		}
		else if (strncmp(user_input,"showfirst",9)==0)
		{
			if (numofelements==0)
			{
				printf("Can't display first 10 elements. Dictionary is empty. \n");
			}
			else if (numofelements > 0)
			{
				printf("\n-------------------------------------------------\n");
				printf("First 10 elements of Dictionary: \n");
				int start = 0;
				int end = (size<10)?size-1:9;
				printDictionary<<<1,1>>>(gpu_output_dict,start,end);
				errors = hipDeviceSynchronize();
				if (errors!=hipSuccess)
				{
					printf("Error in printing first 10 elements of array. ");
				}
				printf("\n-------------------------------------------------\n");
		
				}
			}
		else if (strncmp(user_input,"search",6)==0)
		{
			if (numofelements==0)
			{
				printf("Can't do search operation. Dictionary is empty. \n");
			}
			else
			{
				printf("Search operation. \n");
				hipEventRecord(start,0);
				
				searchDictionary();
				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time,start,stop);
				printf("Time taken for Lookup operation on GPU: %fms.\n",time/1000);
			}
			
		}
		else if (strncmp(user_input,"range-search",12)==0)
		{
			if (numofelements==0)
			{
				printf("Can't do range-search operation. Dictionary is empty. \n");
			}
			else
			{
				printf("Range-Search operation. \n");
				hipEventRecord(start,0);
				rangeQuery();
				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time,start,stop);
				printf("Time taken for Range search/Count operation on GPU: %fms.\n",time/1000);
				
			}
			
		}
		else if (strncmp(user_input,"exit",4)!=0)
		{
			printf("Invalid input. Enter valid input. \n");
			
		}	 

	
	}while(strncmp(user_input,"exit",4)!=0);
	
	errors = hipFree(gpu_output_dict);
	
	if(errors!=hipSuccess)
	{
		fprintf(stderr,"hipFree failed for gpu_output_dict: %s\n",hipGetErrorString(errors));
	}

	
	return 0;
	
}
