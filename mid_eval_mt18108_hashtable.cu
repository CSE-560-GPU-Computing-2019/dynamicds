#include "hip/hip_runtime.h"

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#define MAX_VAL 32
#define MIN_VAL 0
typedef struct Slab Slab;
typedef struct SlabList SlabList;

//Slab is a linkedlist nodes having key value pair
struct Slab {
	Slab **next;
  	int val;
  	int key;
};
//List of ll nodes
struct SlabList {
	Slab *head;
};

SlabList *createSlablist(void);
__device__ void insertSlablist(SlabList *slablist, int ele);
//Method to search for the element befroe insertion
__device__ Slab *node_search(SlabList *slablist, int ele, int search_key);
//Method to create a new node
__device__ Slab *node_create(int val, int key);
//Method to create head node
__global__ void create_head(SlabList *slablist);

__global__ void create_head(SlabList *slablist){
	//Call headnode creation with MAX permitted values
	slablist->head = node_create(MIN_VAL, MAX_VAL);
	memset(slablist->head->next, 0, MAX_VAL * sizeof(Slab *));
}

SlabList *createSlablist(void){
	SlabList *slablist;
	hipMalloc(&slablist, sizeof(SlabList));
	//Invoke a kernel with single thread to create a head node
	create_head<<<1, 1>>>(slablist);
	hipDeviceSynchronize();
	return slablist;
}
//We need to have two reads for synchronize
__device__ void insertSlablist(SlabList *slablist, int ele){
	Slab *new_node, *dest, *read1, *read2;
  	int i, key=1; 
 	while (key < MAX_VAL)
    		key++;// Randomly assign keys
	new_node = node_create(ele, key);
  	for (i = 0; i < key; i++) {
    		do {
      			dest = node_search(slablist, ele, i); // want to insert right after this node
      			read1 = dest->next[i];
      			new_node->next[i] = read1;
	//Ref: From stackoverflow for atomicCAS
      			read2= (Slab *)atomicCAS((unsigned long long int *)&(dest->next[i]),
        *(unsigned long long int *)&read1,
        *(unsigned long long int *)&new_node);
		} while (read1 != read2);
  	}
}
__device__ Slab *node_create(int val, int key){
	Slab *node= (Slab *)malloc(sizeof(Slab));
	node->val = val;
	node->key = key;
	node->next = (Slab **)malloc(key * sizeof(Slab *));
  	return node;
}
//Search for the element before the insertion
__device__ Slab *node_search(SlabList *slablist, int ele, int search_key){
	Slab *cur = slablist->head;
	Slab *next_node;
	int key, flag=0;
	for (key = MAX_VAL - 1;key >= search_key; key--) {
		next_node= cur->next[key];
		while (next_node!= NULL && next_node->val < ele) {
			if(!flag && blockIdx.x==0 && gridDim.x==32 && threadIdx.x==31){
		//		printf("Inserting Elements!!"); 
			printf("ele:%d\n",cur->next[key]->val);	
			}
		cur = next_node;
      		next_node= cur->next[key];
    		}
		flag=1;
  	}
	//if(blockIdx.x==0 && gridDim.x==32 && threadIdx.x==31)
 	printf("_______________");
  	return cur;
}

__global__ void insertKernel(SlabList *slablist, int *a, int N){
	int thId = threadIdx.x + blockIdx.x * blockDim.x;
	while (thId < N) {
    		insertSlablist(slablist, a[thId]);
    		thId += blockDim.x * gridDim.x;
  	}
}

int main(void){
	int N = 32;
  	int *array = (int *)malloc(N * sizeof(int));
  	int *device_array;
  	int i;
  	SlabList *slablist;
  	hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
  	size_t limit;
  	hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
  	hipMalloc(&device_array, N * sizeof(int));
  	for (i = 0; i < N; i++)
    		array[i] = i;
  	printf("done initializing\n");
  	slablist = createSlablist();
	const clock_t begin_time = clock();
  	hipMemcpy(device_array, array, N * sizeof(int), hipMemcpyHostToDevice);
 	insertKernel<<<32, 32>>>(slablist, device_array, N);
  	hipDeviceSynchronize();
	float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
        printf("Time for matching keywords: %fs\n\n", runTime);
  	printf("done inserting.\n");
	return 0;
}
