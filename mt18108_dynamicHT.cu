#include "hip/hip_runtime.h"
//Sujay Raj - MT18108
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <time.h>

typedef struct SlabList SlabList;

#define SLAB_SIZE 33
struct SlabList {
	int val[SLAB_SIZE - 2];
	int key[SLAB_SIZE - 2];
	struct SlabList * next;
};
__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int * lock) {
    while (atomicCAS((int * ) lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int * lock) {
	* lock = 0;
	__threadfence();
}
struct SlabList * createSlablist(struct SlabList * head_ref, int * new_key, int * new_val, int size, int * del_array, int del_size) {
    head_ref = NULL;
    for (int i = 0; i < size / SLAB_SIZE; i++) {
        struct SlabList * new_node = (struct SlabList * ) malloc(sizeof(struct SlabList));
        for (int j = 0; j < SLAB_SIZE; j++) {
            new_node->key[j] = new_key[i * SLAB_SIZE + j];
            new_node->val[j] = new_val[i * SLAB_SIZE + j];
            //printf("key--->%d\tVal---->%d\n",new_node->key[j],new_node->val[j]);
        }
        new_node->next = head_ref;
        head_ref = new_node;
    }

    //Deletion
    //First search for the key and then fill the key and value with #
    while (head_ref != NULL) {
        for (int i = 0; i < SLAB_SIZE; i++) {
            for (int k = 0; k < del_size; k++) {
                if (head_ref->key[i] == del_array[k] && head_ref->key[i] != -999999 && head_ref->key[i] != 0) {
                    //					printf("Found!!! Key: %d\tValue:%d\n",head_ref->key[i],head_ref->val[i]); 		
                    head_ref->key[i] = -999999;
                    head_ref->val[i] = -999999;
                }
            }
        }
        head_ref = head_ref->next;
    }

    return head_ref;
}
void printList(struct SlabList * node) {
    while (node != NULL) {
        for (int i = 0; i < SLAB_SIZE; i++) {
            printf("Key: %d\tValue:%d\n", node->key[i], node->val[i]);
        }
        node = node->next;
    }
}
void printList1(struct SlabList * node, int size) {
    for (int j = 0; j < size; j++) {
        for (int i = 0; i < SLAB_SIZE; i++) {
            printf("Key: %d\tValue:%d\n", node[j].key[i], node[j].val[i]);
        }
    }
}
__global__ void kernelOps(struct SlabList * head_ref, int * new_key, int * new_val, int size, int * del_key, int del_size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        head_ref = NULL;
        struct SlabList * new_node;
        if (id % SLAB_SIZE == 0)//Only one thread per group initialises a SlabList
            new_node = (struct SlabList * ) malloc(sizeof(struct SlabList));
        __syncthreads();
        if ((((id + 1) % SLAB_SIZE)) != 0) { //31,63,95 etc.,
            new_node[id / SLAB_SIZE].key[id] = new_key[id]; //Insert values to new_node[0],new_node[1] etc.,
            new_node[id / SLAB_SIZE].val[id] = new_val[id];
            //			printf(":key--->%d\tVal---->%d\n",new_node[id/SLAB_SIZE].key[id],new_node[id/SLAB_SIZE].val[id]);
        }
        if (id % SLAB_SIZE == SLAB_SIZE-1){ //All last nodes in the warp has to update the next counter
		 new_node[id / SLAB_SIZE].next = & (new_node[id / SLAB_SIZE + 1]);
        }
	__syncthreads();
        new_node->next = head_ref;
        //if (threadIdx.x == 0)
        //acquire_semaphore(&sem);	
        //__syncthreads();
        //memcpy(SL,new_node, size * sizeof(struct SlabList));
        
	//One node to search for element
//        if ((id % SLAB_SIZE) == 0) { //First thread of each warp searches for key in its Slab
	if(id<del_size){ 
      //    for (int k = 0; k < del_size; k++) {
            	for (int j = 0; j < SLAB_SIZE - 1; j++) {
           	    if(__shfl(new_node[id/SLAB_SIZE].key[j],(id+1)%SLAB_SIZE,32)==del_key[id] && __shfl(new_node[id/SLAB_SIZE].key[j],(id+1)%SLAB_SIZE,32) == -999999) {
                    //if (new_node[id / SLAB_SIZE].key[j] == del_key[k] && new_node[id / SLAB_SIZE].key[j] == -999999) {
                        //printf("found!!!\n");
			//Delete the node
                        new_node[id / SLAB_SIZE].key[j] = -999999;
                        new_node[id / SLAB_SIZE].val[j] = -999999;
                    }
                }
            //}
        }
        //__syncthreads();
        //if (threadIdx.x == 0)
        //release_semaphore(&sem);
        //__syncthreads();
    }
}

int main(int argc, char** argv) {
    int N = 1000000, M = 1000; //N: Insert Size; M: Del Size
    if(argc==2)
	N=atoi(argv[1]);
    if(argc==3){
	N= atoi(argv[1]);
	M= atoi(argv[2]);
}
/*if(M>N){
	printf("Not possible to delete!!!(M>N)\n");
	exit(0);
}*/
printf("----------------\nN: %d\tM:%d\n",N,M);
    int * val_array = (int * ) malloc(N * sizeof(int));
    int * key_array = (int * ) malloc(N * sizeof(int));
    int * del_key_array = (int * ) malloc(M * sizeof(int));
    int * d_val_array = NULL;
    int * d_key_array = NULL;
    int * d_del_key_array = NULL;
    float kTime=0, kplusMTime=0;
    struct SlabList * start = (struct SlabList * ) malloc(sizeof(struct SlabList));
    struct SlabList * d_start = NULL;
    hipMalloc( & d_start, N * sizeof(struct SlabList));
    hipMalloc( & d_val_array, N * sizeof(int));
    hipMalloc( & d_key_array, N * sizeof(int));
    for (int i = 0; i < N; i++) {
        val_array[i] = i;
        key_array[i] = i + 10;
    }
    //Fill random with del_key array
    srand(time(0));
    for (int i = 0; i < M; i++) {
        int r = rand() % N;
        del_key_array[i] = r;
    }
    const clock_t seq_begin_time = clock();
    //Batch insertion
    start = NULL;
    struct SlabList * head = createSlablist(start, val_array, key_array, N, del_key_array, M);
    //	printList(head);
    float seq_runTime = (float)(clock() - seq_begin_time) / CLOCKS_PER_SEC;
    printf("Seq Time for matching keywords: %fs\n\n", seq_runTime);
    const clock_t par_begin_time1 = clock();
    //printf("done initializing\n");
    hipMemcpy(d_val_array, val_array, N * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(struct SlabList) * N);
    hipMemcpy(d_key_array, key_array, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_del_key_array, del_key_array, M * sizeof(int), hipMemcpyHostToDevice);
    const clock_t par_begin_time2 = clock();
    int it=100;
    for(int i=0;i<it;i++){
    kernelOps << < 512, 512 >>> (d_start, d_val_array, d_key_array, N, d_del_key_array, M);
    hipDeviceSynchronize();
    float par_runTime2 = (float)(clock() - par_begin_time2) / CLOCKS_PER_SEC;
    struct SlabList * head1 = (struct SlabList * ) malloc(N * sizeof(struct SlabList));
    //hipMemcpy(head1, d_SL, N * sizeof(struct SlabList), hipMemcpyDeviceToHost);
    float par_runTime1 = (float)(clock() - par_begin_time1) / CLOCKS_PER_SEC;
    kTime+=par_runTime2;
    kplusMTime+=par_runTime1;
  //  hipFree(d_val_array);
    //hipFree(d_key_array);
   // hipFree(d_del_key_array); 
}   
    printf("Kernel timing: %fs\n\n", kTime/it);
    printf("Kernel plus memcopy timing: %fs\n\n", kplusMTime/it);
    printf("Speedup Over sequential execution is %f\n",seq_runTime*it/kTime);
//       printf("____________________________GPU Insertion!!!_______________________________________\n");
    //        printList1(head1,N/SLAB_SIZE);
    return 0;
}
