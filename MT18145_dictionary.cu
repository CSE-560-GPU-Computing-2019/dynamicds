//MT18145(Shubham Kumar)

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#define NUM 16
//key-value pairs are also referred to as elements in this code.
//Structure for dictionary user-defined data type
//Use structure of arrays instead to use sort and merge
//Dictionary is not ordered and does not search for duplicate
//keys. It supports any number of elements to be inserted
//while executing on GPU. So it is dynamic dictionary.
//Key value pairs are stored in order in which they are inserted.
typedef struct
{
	char key[100];
	char value[100];
	//char* key;
	//char* value;
}Dictionary;

//batchsize < size(in this project for other operations, not insertion)
//Kernel to insert elements in existing dynamic GPU dictionary(Array of Structures)
__global__ void insertBatch(Dictionary *dictionary, Dictionary *data, int batchsize, int size)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//key_len = strlen(data[index].key);
	//value_len = strlen(data[index].value);
	//for (int i=0;i<key_len;i++)
	if (index<batchsize)
	{
		for (int i=0;i<100;i++)
		{
			dictionary[index+size].key[i] = data[index].key[i];
		}
		//for (int i=0;i<value_len;i++)
		for (int i=0;i<100;i++)
		{
			dictionary[index+size].value[i] = data[index].value[i];
		}	
	}
	 
}
//Kernel to copy key-value pairs of two different Dictionary Array of Structures
__global__ void copyongpu(Dictionary* dst,Dictionary* src, int batchsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index<batchsize)
	{
		for (int i=0;i<100;i++)
		{
			dst[index].key[i] = src[index].key[i];
			dst[index].value[i] = src[index].value[i];	
		}
		
	}
}

int main()
{
	int num = NUM, blocksize, numofblocks;
	//num represents number of elements user wants to enter
	//in any iteration
	//blocksize, numofblocks are used for kernel launch
	char input_string[200];//To take input from user
	int maxelements = 10;
	//if size of array of structure on GPU exceeds maxelements,
	//then array of structure is resized,i.e., maxelements*=2
	//and GPU array of structure is copied to temporary array
	//of structure and again copied back when resizing is done,
	//then insertion is done

	int size = 0;//Number of estimated key-value pairs in 
	//Array of Structure after insertion
	int numofelements = 0;// Number of existing key-value pairs
	char *token = NULL;//Used to separate key and value inserted by user
	printf("Enter number of key-value pairs. Enter 0 to exit. ");
	scanf("%d",&num);
	//If user enters 0 or negative number, error is thrown
	if (num<=0)
	return 0;
	Dictionary *gpu_output_dict;
	//desired Array of Structures on GPU of type Dictionary
	//to store key value pairs on GPU 
	//with inclusion of new key value pairs recently inserted
	hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
	for (int j=0;j<maxelements;j++)
	{
		//limit on word length of key and value <= 100 characters
		hipMemset(gpu_output_dict[j].key,'\0',sizeof(gpu_output_dict[j].key));
		hipMemset(gpu_output_dict[j].value,'\0',sizeof(gpu_output_dict[j].value));
			 
	}
	Dictionary *gpu_temp_dict;
	//Array of Structures of type Dictionary
	//to store elements temporarily so that 
	//gpu_output_dict can be resized
	//Iteration starts
	//User would enter key-value pairs atleast once if num>0 from
	//input taken above
	//Iteration ends when user enters 0 or a negative number
	do
	{
		/*if (size<num)
		{
			//batchsize should always be less than number of existing elements
			printf("Number of elements to be inserted cannot be larger than existing elements. Exiting.");
			break;
		}*/
		size+=num;
		//represents total no of elements after insertion
		//Insertion yet to be done
		//printf("Number of elements: %d\n",numofelements);
		//printf("Number of elements estimated after insertion: %d\n",size);
		
		
		if (size>maxelements)
		{//resizing needed
			int numofblocks2 = 0, blocksize2 = 0;
			//blocksize2, numofblocks2 are used for kernel launch

			maxelements *= 2;
			hipMalloc((void**)&gpu_temp_dict,maxelements*sizeof(Dictionary));
			for (int j=0;j<maxelements;j++)
			{
			//limit on word length of key and value <= 100 characters
				hipMemset(gpu_temp_dict[j].key,'\0',sizeof(gpu_output_dict[j].key));
				hipMemset(gpu_temp_dict[j].value,'\0',sizeof(gpu_output_dict[j].value));
			 
			}
			
			if (maxelements>1024)
			{
				numofblocks2 = (maxelements%1024)?(maxelements/1024):(maxelements/1024+1);
				blocksize2 = 1024;
			}
			else
			{
				numofblocks2 = 1;
				blocksize2 = maxelements;	
			}
			copyongpu<<<numofblocks2,blocksize2>>>(gpu_temp_dict,gpu_output_dict,maxelements);
			hipFree(gpu_output_dict);
			hipMalloc((void**)&gpu_output_dict,maxelements*sizeof(Dictionary));
			
			for (int j=0;j<maxelements;j++)
			{
			//limit on word length of key and value <= 100 characters
				hipMemset(gpu_output_dict[j].key,'\0',sizeof(gpu_output_dict[j].key));
				hipMemset(gpu_output_dict[j].value,'\0',sizeof(gpu_output_dict[j].value));
			 
			}
			copyongpu<<<numofblocks2,blocksize2>>>(gpu_output_dict,gpu_temp_dict,maxelements);
			hipFree(gpu_temp_dict);

		}
		
		if (num>1024)
		{
			numofblocks = (num%1024)?(num/1024):(num/1024+1);
			blocksize = 1024;
		}
		else
		{
			numofblocks = 1;
			blocksize = num;	
		}
		Dictionary *dict = (Dictionary*)malloc(num*sizeof(Dictionary));
		for (int j=0;j<num;j++)
		{//limit on word length of key and value <= 100 characters
			printf("Enter key and value separated by - : ");
			
			scanf("%s",input_string);
			//printf("Input: %s",input_string);
			token = strtok(input_string,"-");
			memset(dict[j].key,'\0',sizeof(dict[j].key));
			memset(dict[j].value,'\0',sizeof(dict[j].value));
			strncpy(dict[j].key,token,100);
			//printf("Key: %s\n",dict[j].key);
			token = strtok(NULL,"-");
			strncpy(dict[j].value,token,100);
			//printf("Value: %s\n",dict[j].value);
			 
			//dict[j].key = (char*)malloc(100);
			//dict[j].value = (char*)malloc(100);
			
			//printf("Enter value: ");
			
		}
		Dictionary *gpu_input_dict;
		hipMalloc((void**)&gpu_input_dict,num*sizeof(Dictionary));
		
		hipMemcpy(gpu_input_dict,dict,num*sizeof(Dictionary),hipMemcpyHostToDevice);
		
		//cudaMemset((void**)&gpu_output_dict,'\0',num*sizeof(Dictionary));
		insertBatch<<<numofblocks,blocksize>>>(gpu_output_dict,gpu_input_dict,num,numofelements);
		Dictionary *cpu_output_dict = (Dictionary*)malloc(maxelements*sizeof(Dictionary));
		//memset(cpu_output_dict,'\0',sizeof(Dictionary)*num);
		for (int j=0;j<maxelements;j++)
		{//limit on word length of key and value <= 100 characters
			memset(cpu_output_dict[j].key,'\0',sizeof(cpu_output_dict[j].key));
			memset(cpu_output_dict[j].value,'\0',sizeof(cpu_output_dict[j].value));
			 
		}
		
		hipMemcpy(cpu_output_dict,gpu_output_dict,maxelements*sizeof(Dictionary),hipMemcpyDeviceToHost);
		printf("\n-------------------------------------------------\n");
		for (int j=0;j<size;j++)
		{
			printf("Key: %s\tValue: %s\n",cpu_output_dict[j].key,cpu_output_dict[j].value);
		}
		printf("-------------------------------------------------\n");
		free(dict);
		free(cpu_output_dict);
		hipFree(gpu_input_dict);
		numofelements+=num;
		printf("Enter number of key-value pairs. Enter 0 to exit. ");
		scanf("%d",&num);
		
	}while (num>0);
	hipFree(gpu_output_dict);

}
