//MT18145(Shubham Kumar)

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#define NUM 16
//Structure for dictionary user-defined data type
typedef struct
{
	char key[100];
	char value[100];
	//char* key;
	//char* value;
}Dictionary;


__global__ void insertBatch(Dictionary *dictionary, Dictionary *data)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//key_len = strlen(data[index].key);
	//value_len = strlen(data[index].value);
	//for (int i=0;i<key_len;i++)
	for (int i=0;i<100;i++)
	{
		dictionary[index].key[i] = data[index].key[i];
	}
	//for (int i=0;i<value_len;i++)
	for (int i=0;i<100;i++)
	{
		dictionary[index].value[i] = data[index].value[i];
	} 
}

int main()
{
	int num = NUM, blocksize, numofblocks;
	char *input_string = (char*)malloc(sizeof(char)*200);
	printf("Enter number of key-value pairs. Enter 0 to exit. ");
	scanf("%d",&num);
	if (num<=0)
	return 0;
	do
	{
		if (num>1024)
		{
			numofblocks = (num%1024)?(num/1024):(num/1024+1);
			blocksize = 1024;
		}
		else
		{
			numofblocks = 1;
			blocksize = num;	
		}
		Dictionary *dict = (Dictionary*)malloc(num*sizeof(Dictionary));
		for (int j=0;j<num;j++)
		{//limit on word length of key and value <= 100 characters
			printf("Enter key and value separated by - : ");
			fgets(input_string,200,stdin);
			char *token = strtok(input_string,"-");
			strncpy(dict[j].key,token,sizeof(dict[j].key));
			token = strtok(input_string,"-");
			strncpy(dict[j].value,token,sizeof(dict[j].value));
			 
			//dict[j].key = (char*)malloc(100);
			//dict[j].value = (char*)malloc(100);
			//gets(dict[j].key);
			//printf("Enter value: ");
			//gets(dict[j].value);
			//fgets(dict[j].value,100,stdin);
		}
		Dictionary *gpu_output_dict,*gpu_input_dict;
		hipMalloc((void**)&gpu_input_dict,num*sizeof(Dictionary));
		
		hipMemcpy(gpu_input_dict,dict,num*sizeof(Dictionary),hipMemcpyHostToDevice);
		hipMalloc((void**)&gpu_output_dict,num*sizeof(Dictionary));
		hipMemset((void**)&gpu_output_dict,'\0',num*sizeof(Dictionary));
		insertBatch<<<numofblocks,blocksize>>>(gpu_output_dict,gpu_input_dict);
		Dictionary *cpu_output_dict = (Dictionary*)malloc(num*sizeof(Dictionary));
		memset(cpu_output_dict,'\0',sizeof(Dictionary)*num);
		hipMemcpy(cpu_output_dict,gpu_output_dict,num*sizeof(Dictionary),hipMemcpyDeviceToHost);
		
		for (int j=0;j<num;j++)
		{
			printf("Key: %s\tValue: %s\n",cpu_output_dict[j].key,cpu_output_dict[j].value);
		}
		printf("Enter number of key-value pairs. Enter 0 to exit. ");
		scanf("%d",&num);
		
	}while (num>0);
}