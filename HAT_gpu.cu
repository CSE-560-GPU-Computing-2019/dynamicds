
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define size 100


void print_tree(int ** HAT, int n){
	for (int i = 0; i < n; i++){
		if (HAT[i] != NULL){
			printf("\n");
			for (int j =0; j<n ; j++){
				printf("%d ",HAT[i][j]);
			}
		}
	}
}


/*
void insert_tree (int ** HAT, int n, int * input, int inputsize) {
	for (int i = 0; i < inputsize; i++){
		int j = i/n;
		if (HAT[j]==NULL){
			HAT[j] = malloc(sizeof(int) * n);
		}
		int k = i % n;
		HAT[j][k] = input[i];
	}
}
*/

__global__ void insert_gpu (int ** HAT, int *input, int inputsize, int n) {
	printf("Inserted");
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	
	if(i<inputsize){
		printf("Inserted");
		int j = i/n;
		int k = i % n;
		HAT[j][k] = input[i];
	}
}


int main (int argc, const char **argv) {
	printf("1");
	int ** HAT;
	printf("2");
	int n = (int)sqrt(size);
	printf("sdfsdv");
	HAT = (int **)malloc(sizeof(int *) * n);
	for (int i =0; i<n; i++){
		HAT[i] = (int *)malloc(sizeof(int) * n);
	}
	//HAT[0] = malloc(sizeof(int) * n);
	//HAT[0][1] = 1;
	//printf("3");
	//printf("%d",HAT[0][1]);
	//printf("%d",HAT[0][2]);
	//printf("%d",HAT[0][3]);
	//printf("%d",HAT[1][1]);
	
	int input[100];
	int inputsize = 100;
	for (int i =0; i<inputsize; i++) {
		input[i] = i;
	}
	printf("3");
	//GPU code starts here
	int * input_d;
	int ** HAT_d;
	hipMalloc ((void **)&input_d , sizeof(int) * inputsize);
	hipMalloc ((void **)HAT_d , sizeof(int *) * n);
	
	for (int i =0; i<n; i++){
		hipMalloc((void**)&HAT_d[i],sizeof(int) * n);
	}
	
	hipMemcpy (input_d, input, sizeof(int) * inputsize , hipMemcpyHostToDevice);
	
	int no_of_blocks;
	
	if (inputsize % 1024 == 0)
		no_of_blocks = inputsize/1024;
	else
		no_of_blocks = (inputsize/1024) + 1;
	printf("4");
	insert_gpu<<<no_of_blocks,1024 >>>(HAT_d,input_d, inputsize, n);
	printf("5");
	//insert_tree(HAT, n , input, inputsize);
	
	
	hipMemcpy (HAT, HAT_d, sizeof(int*) * n , hipMemcpyDeviceToHost);
	print_tree(HAT,n);
	hipFree(HAT_d);
	hipFree(input_d);
	return 0;
}
