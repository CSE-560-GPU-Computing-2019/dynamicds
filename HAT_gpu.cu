#include "hip/hip_runtime.h"
//Krishna Bagaria MT18128

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define size 100


void print_tree(int ** HAT, int n){
	for (int i = 0; i < n; i++){
		if (HAT[i] != NULL){
			printf("\nBucket %d : ",i);
			for (int j =0; j<n ; j++){
				printf("%d ",HAT[i][j]);
			}
		}
	}
}


void insert_tree (int ** HAT, int n, int * input, int inputsize) {
	for (int i = 0; i < inputsize; i++){
		int j = i/n;
		if (HAT[j]== NULL){
			HAT[j] = (int *)malloc(sizeof(int) * n);
		}
		int k = i % n;
		HAT[j][k] = input[i];
	}
}

__global__ void insert_gpu (int ** HAT_d, int *input, int inputsize, int n) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x ;
	
	if(i<inputsize){
		
		int j = i/n;
		int k = i % n;
		HAT_d[j][k] = input[i];
		
	}
}


int main (int argc, const char **argv) {
	int ** HAT;
	int n = sqrt(size);
	HAT = (int **)malloc(sizeof(int *) * n);

	int inputsize;
	printf("Enter no. of elements to be inserted: ");
	scanf("%d",&inputsize);
	
	int * input = (int *)malloc(sizeof(int) * inputsize);
	printf("Enter the elements (integers): ");
	
	for (int i =0; i<inputsize; i++) {
		scanf("%d", &input[i]);
	}
	
	const clock_t begin_time = clock();
	insert_tree(HAT, n , input, inputsize);
	float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
	printf("Time for inserting(CPU): %fs\n\n", runTime);
	printf("\nOutput Tree by inserting from CPU:\n");
	print_tree(HAT,n);
	
	//GPU code starts here
	int * input_d;
	int ** HAT_d;
	
	hipMalloc ((void **)&input_d , sizeof(int) * inputsize);
	hipMalloc ((void ***)&HAT_d , sizeof(int *) * n);
	
	int **support;
	support = (int**) malloc(sizeof(int*)*n);
	hipMemcpy(support,HAT_d,n*sizeof(int*),hipMemcpyDeviceToHost);
	
	for (int i =0; i<n; i++){
		hipMalloc((void**)&support[i],sizeof(int) * n);
	}
	
	hipMemcpy (input_d, input, sizeof(int) * inputsize , hipMemcpyHostToDevice);
	
	int grid_size = (inputsize % 1024) ? ((inputsize/1024) + 1) : (inputsize/1024);
	int block_size = 1024;
	
	const clock_t begin_time1 = clock();
	insert_gpu<<<grid_size,block_size>>>(HAT_d,input_d, inputsize, n);
	hipDeviceSynchronize();
	runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
	printf("\n\nTime for inserting(GPU): %fs\n\n", runTime);
	
	hipMemcpy (HAT, HAT_d, sizeof(int*) * n , hipMemcpyDeviceToHost);
	
	printf("\nOutput Tree by inserting from GPU:\n");
	print_tree(HAT,n);
	
	hipFree(HAT_d);
	hipFree(input_d);
	free(HAT);
	free(input);
	
	return 0;
}
